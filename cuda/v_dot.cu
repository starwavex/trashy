
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#define DIM_V 1000

__global__ void cu_add(double *d1, double *d2, double *d3, unsigned N) {
	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		d3[i] = d1[i] + d2[i];
}

__global__ void cu_scl(double scl, double *d1, double *d2, unsigned N) {
	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		d2[i] = scl * d1[i];
}

__global__ void cu_func(double *d1, double *d2, unsigned N) {
	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		d2[i] = sin(d1[i])/d1[i];
}

__global__ void cu_razb(double a, double d, double* x, unsigned N) {
	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		x[i] = a + d * ((double)i + 1./2.);
}

__global__ void cu_sum(double *d_dst, double *d_src, unsigned N, unsigned N_ofElements) {
	unsigned element, destination;
	
	destination = blockDim.x * blockIdx.x + threadIdx.x;
	d_dst[destination] = 0.;

	for (unsigned k = 0; k < N_ofElements; ++k) {
		element = destination * N_ofElements + k;
		if (element < N) {
			d_dst[destination] += d_src[element];
		}
	}
}

double e_sum(double *x, unsigned N) {
	unsigned threadsPerBlock, blocksPerGrid, sizeOfPart, sizeOfSum;
	size_t x_size, sum_size;
	double *d_x, *d_sum;
	double sum;

	sizeOfPart = (unsigned)floor(sqrt((double)N));
	sizeOfSum = (unsigned)ceil( ((double)N) / ((double)sizeOfPart) );
	blocksPerGrid = (unsigned)ceil( ((double)sizeOfSum) / ((double)1024) );
	threadsPerBlock = (sizeOfSum < 1024) ? sizeOfSum : 1024;

	//printf("sizeOfPart = %u\nsizeOfSum = %u\nblocksPerGrid = %u\nthreadsPerBlock = %u\n", sizeOfPart, sizeOfSum, blocksPerGrid, threadsPerBlock);
	x_size = N * sizeof(double);
	sum_size = sizeOfSum * sizeof(double);

	hipMalloc(&d_x, x_size);
	hipMalloc(&d_sum, sum_size);

	hipMemcpy(d_x, x, x_size, hipMemcpyHostToDevice);
	
	cu_sum<<<blocksPerGrid, threadsPerBlock>>>(d_sum, d_x, N, sizeOfPart);

	hipFree(d_x);
	hipMalloc(&d_x, sizeof(double));
	cu_sum<<<1,1>>>(d_x, d_sum, sizeOfSum, sizeOfSum);

	hipMemcpy(&sum, d_x, sizeof(double), hipMemcpyDeviceToHost);
	

	hipFree(d_x);
	/*
	d_x = (double*)malloc(sum_size);
	cudaMemcpy(d_x, d_sum, sum_size, cudaMemcpyDeviceToHost);
	sum = 0.;
	for (unsigned i = 0; i < sizeOfSum; ++i) {
		sum += d_x[i];
	}
	free(d_x);
	*/
	hipFree(d_sum);

	return sum;
}

double ntgrl(double a, double b, unsigned N) {
	double d, I;
	double *x;
	size_t size;

	d = (b - a)/(double)N;
	size = N * sizeof(double);

	hipMalloc(&x, size);
	
	unsigned threadsPerBlock = 1024;
	unsigned blocksPerGrid = (N - 1) / threadsPerBlock + 1;

	cu_razb<<<blocksPerGrid, threadsPerBlock>>>(a, d, x, N);
	cu_func<<<blocksPerGrid, threadsPerBlock>>>(x, x, N);
	cu_scl<<<blocksPerGrid, threadsPerBlock>>>(d, x, x, N);
	
	I = e_sum(x, N);
	hipFree(x);
	return I;
}

int main(void) {
	double I, a, b;
	unsigned N;
	printf("Enter a: "); scanf("%lf", &a);
	printf("Enter b: "); scanf("%lf", &b);
	printf("Enter precision N: "); scanf("%u", &N);
	I = ntgrl(a, b, N);
	printf("Ntgrl = %lf\n", I);
	return 0;
}
